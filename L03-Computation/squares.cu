#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 100

__global__ void dsq(int *a) {
    int i = threadIdx.x;
    a[i] = i*i;
}

int main() {
    int a[N], *da;
    
    hipMalloc(&da, N * sizeof(int));
    
    dsq<<<1,N>>>(da);
    hipMemcpy(a, da, N * sizeof(int), hipMemcpyDeviceToHost);
    
    for (int i = 0; i < N; i++) {
        printf("%d\n", a[i]);
    }
    return 0;
}