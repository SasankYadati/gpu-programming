#include<stdio.h>
#include<hip/hip_runtime.h>

void squarecpu(unsigned *matrix, unsigned *result, unsigned matrixsize) {
    for (unsigned ii=0; ii<matrixsize; ii++) {
        for (unsigned jj=0; jj<matrixsize; jj++) {
            for (unsigned kk=0; kk<matrixsize; kk++) {
                result[ii*matrixsize + jj] += matrix[ii * matrixsize + kk] * matrix[kk * matrixsize + jj];
            }
        }
    }
}

__global__ void squaregpu(unsigned *matrix, unsigned *result, unsigned matrixsize) {
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    for (unsigned jj = 0; jj<matrixsize; jj++) {
        for (unsigned kk = 0; kk<matrixsize; kk++) {
            result[id*matrixsize+kk] += matrix[id * matrixsize + kk] * matrix[kk * matrixsize + jj];
        }
    }
}

__global__ void squaregpu2(unsigned *matrix, unsigned *result, unsigned matrixsize) {
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned ii = id / matrixsize;
    unsigned jj = id % matrixsize;
    for (unsigned kk = 0; kk < matrixsize; kk++) {
        result[ii * matrixsize + jj] += matrix[ii * matrixsize + kk] * matrix[kk * matrixsize + jj];
    }
}

int main() {
    unsigned *matrix,*result;
    int matrixsize = 64;
    squaregpu<<<1,matrixsize>>>(matrix, result, matrixsize);
    squaregpu2<<<matrixsize, matrixsize>>>(matrix, result, matrixsize);
    hipDeviceSynchronize();
    return 0;
}