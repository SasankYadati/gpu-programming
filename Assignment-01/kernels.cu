#include "hip/hip_runtime.h"
#include "kernels.h"
#include <hip/hip_runtime.h>

__global__ void per_row_kernel(int m,int n,int *A,int *B,int *C) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i=0; i<n; i++) {
        C[id * m + i] = A[id * m + i] + B[id * m + i];
    }
}

__global__ void per_column_kernel(int m,int n,int *A,int *B,int *C) {

}

__global__ void per_element_kernel(int m,int n,int *A,int *B,int *C) {
    
}
