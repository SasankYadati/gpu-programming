#include "hip/hip_runtime.h"
#include "kernels.h"
#include <hip/hip_runtime.h>

__global__ void per_row_kernel(int m,int n,int *A,int *B,int *C) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < m) {
        for (int i=0; i<n; i++) {
            C[id * n + i] = A[id * n + i] + B[id * n + i];
        }
    }
}

__global__ void per_column_kernel(int m,int n,int *A,int *B,int *C) {
    int x = threadIdx.x;
    int y = threadIdx.y;
    int id = blockIdx.x * blockDim.x * blockDim.y + (x * blockDim.x + y);
    if (id < n) {
        for (int i=0; i<m; i++) {
            C[i * n + id] = A[i * n + id] + B[i * n + id];
        }
    }
}

__global__ void per_element_kernel(int m,int n,int *A,int *B,int *C) {
    int x = blockIdx.x;
    int y = blockIdx.y;
    int blockId = x * gridDim.x + y;
    int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.x * blockDim.x + threadIdx.y);
    int id = threadId;
    if (id < m*n) {
        C[id] = A[id] + B[id];
    }
}
