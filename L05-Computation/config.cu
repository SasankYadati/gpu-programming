#include<stdio.h>
#include<hip/hip_runtime.h>
#define BLOCKSIZE 1024

__global__ void dkernel(unsigned *matrix, int len) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < len) {
        matrix[id] = id;
    }
}

int main(int nn, char *str[]) {
    unsigned N = atoi(str[1]);
    unsigned *vector, *hvector;
    hipMalloc(&vector, N * sizeof(unsigned));
    hvector = (unsigned *)(malloc(N*sizeof(int)));

    unsigned nblocks = ceil((float)N / BLOCKSIZE);
    printf("nblocks = %d\n", nblocks);

    dkernel<<<nblocks, BLOCKSIZE>>>(vector, N);
    hipMemcpy(hvector, vector, N*sizeof(unsigned), hipMemcpyDeviceToHost);
    for (unsigned ii=0; ii<N; ii++) {
        printf("%4d ", hvector[ii]);
    }
    return 0;
}