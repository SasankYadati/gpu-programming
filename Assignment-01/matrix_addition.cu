#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "kernels.h"
#define BLOCKSIZE 1024

using namespace std;

void readMatrix(int *matrix, int count);

int main() {
    int m,n;
    cin>>m>>n;
    int *hmatrixA, *hmatrixB, *hmatrixC;
    hmatrixA = (int*)(malloc(m*n*sizeof(int)));
    hmatrixB = (int*)(malloc(m*n*sizeof(int)));
    hmatrixC = (int*)(malloc(m*n*sizeof(int)));
    readMatrix(hmatrixA, m*n);
    readMatrix(hmatrixB, m*n);
    
    printf("--------------------\n");
    for (int i=0; i<m; i++) {
        for (int j=0; j<n; j++) {
            printf("%d ", hmatrixA[i * n + j]);
        }
        printf("\n");
    }

    printf("--------------------\n");
    for (int i=0; i<m; i++) {
        for (int j=0; j<n; j++) {
            printf("%d ", hmatrixA[i * n + j]);
        }
        printf("\n");
    }

    int *matrixA, *matrixB, *matrixC;
    hipMalloc(&matrixA, m*n*sizeof(int));
    hipMalloc(&matrixB, m*n*sizeof(int));
    hipMalloc(&matrixC, m*n*sizeof(int));
    
    hipMemcpy(matrixA, hmatrixA, m*n*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(matrixB, hmatrixB, m*n*sizeof(int), hipMemcpyHostToDevice);

    int num_blocks = ceil((float)m/BLOCKSIZE);

    per_row_kernel<<<num_blocks,BLOCKSIZE>>>(m, n, matrixA, matrixB, matrixC);

    hipMemcpy(hmatrixC, matrixC, m*n*sizeof(int), hipMemcpyDeviceToHost);

    printf("--------------------\n");
    for (int i=0; i<m; i++) {
        for (int j=0; j<n; j++) {
            printf("%d ", hmatrixC[i * n + j]);
        }
        printf("\n");
    }


    num_blocks = ceil((float)n/(BLOCKSIZE*BLOCKSIZE));
    dim3 blockDims(BLOCKSIZE, BLOCKSIZE);

    per_column_kernel<<<num_blocks, blockDims>>>(m, n, matrixA, matrixB, matrixC);
    
    hipMemcpy(hmatrixC, matrixC, m*n*sizeof(int), hipMemcpyDeviceToHost);

    printf("--------------------\n");
    for (int i=0; i<m; i++) {
        for (int j=0; j<n; j++) {
            printf("%d ", hmatrixC[i * n + j]);
        }
        printf("\n");
    }

    num_blocks = ceil((float)(m*n)/(BLOCKSIZE*BLOCKSIZE));
    dim3 gridDims(num_blocks/2, num_blocks-(num_blocks/2)+1);

    per_element_kernel<<<gridDims, blockDims>>>(m, n, matrixA, matrixB, matrixC);

    hipMemcpy(hmatrixC, matrixC, m*n*sizeof(int), hipMemcpyDeviceToHost);

    printf("--------------------\n");
    for (int i=0; i<m; i++) {
        for (int j=0; j<n; j++) {
            printf("%d ", hmatrixC[i * n + j]);
        }
        printf("\n");
    }

    return 0;
}

void readMatrix(int *matrix, int count) {
    int elem;
    for (int i=0; i<count; i++) {
        cin>>elem;
        matrix[i] = elem;
    }
}