#include<iostream>
#include<hip/hip_runtime.h>
#include <cmath>
#define BLOCKSIZE 1024

using namespace std;

int getNumBlocks(dim3 gridDim) {
    return gridDim.x * gridDim.y * gridDim.z;
}

int getNumThreadsPerBlock(dim3 blockDim) {
    return blockDim.x * blockDim.y * blockDim.z;
}

__global__ void dkernel(int *vectorX, int *vectorY, float *dist, int N) {
    int id1 = threadIdx.x, id2 = threadIdx.y;
    id1 += blockDim.x * blockIdx.x;
    id2 += blockDim.y * blockIdx.y;
    if (id1<id2) {
        auto x1 = vectorX[id1], y1 = vectorY[id1], x2 = vectorX[id2], y2 = vectorY[id2];
        dist[id1 * blockDim.x + id2] = sqrt((float)(pow(x1 - x2, 2) + pow(y1 - y2, 2)));
        dist[id2 * blockDim.x + id1] = dist[id1 * blockDim.x + id2];
    }
}

void readVectors(int *vec1, int *vec2, int count);


int main(int nn, char *str[]) {
    unsigned N = atoi(str[1]);

    int nthreads = N * N;
    int nthreads_per_block_x =  nthreads < BLOCKSIZE ? N : BLOCKSIZE;
    int nthreads_per_block_y = nthreads_per_block_x;

    dim3 blockdims(nthreads_per_block_x,nthreads_per_block_y,1);
    unsigned nblocks = ceil((float)(nthreads) / (nthreads_per_block_x * nthreads_per_block_y));
    printf("nblocks = %d\n nthreads=%d\n blockdims=(%d, %d)\n", nblocks, nthreads, nthreads_per_block_x, nthreads_per_block_y);

    int *vectorX, *vectorY, *hvectorX, *hvectorY;
    
    hipMalloc(&vectorX, N * sizeof(int));
    hipMalloc(&vectorY, N * sizeof(int));
    hvectorX = (int *)(malloc(N*sizeof(int)));
    hvectorY = (int *)(malloc(N*sizeof(int)));
    
    readVectors(hvectorX, hvectorY, N);
    hipMemcpy(vectorX, hvectorX, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(vectorY, hvectorY, N*sizeof(int), hipMemcpyHostToDevice);

    float *dist, *hdist;
    hipMalloc(&dist, N * N * sizeof(float));
    hdist = (float *)(malloc(N * N * sizeof(float)));

    dkernel<<<nblocks,blockdims>>>(vectorX, vectorY, dist, N);
    hipMemcpy(hdist, dist, N * N * sizeof(float), hipMemcpyDeviceToHost);

    for (unsigned ii=0; ii<N; ii++) {
        for (unsigned jj=0; jj<N; jj++) {
            printf("%5.2f ", hdist[ii * N + jj]);
        }
        printf("\n");
    }

    return 0;
}

void readVectors(int *vec1, int *vec2, int count) {
    int x,y;
    for (int i=0; i<count; i++) {
        cin>>x>>y;;
        vec1[i] = x;
        vec2[i] = y;
    }
}