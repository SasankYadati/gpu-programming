#include<stdio.h>
#include<hip/hip_runtime.h>
#define N 5
#define M 6

__global__ void dkernel(unsigned *matrix) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    matrix[id] = id;
}

int main() {
    unsigned *matrix, *hmatrix;
    hipMalloc(&matrix, N*M*sizeof(unsigned));
    hmatrix = (unsigned *)(malloc(N*M*sizeof(unsigned)));

    dkernel<<<N,M>>>(matrix);
    hipMemcpy(hmatrix, matrix, N*M*sizeof(unsigned), hipMemcpyDeviceToHost);

    for (unsigned ii=0; ii<N; ii++) {
        for (unsigned jj=0; jj<M; jj++) {
            printf("%2d ", hmatrix[ii * M + jj]);
        }
        printf("\n");
    }

    return 0;    
}