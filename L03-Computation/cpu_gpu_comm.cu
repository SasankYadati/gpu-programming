#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dkernel(char *arr, int arrlen) {
    unsigned id = threadIdx.x;
    if (id < arrlen) {
        ++arr[id];
    }
}

int main() {
    char cpuarr[] = "Gdn Vnc-";
    char *gpuarr;

    auto sz = sizeof(char) * (1 + strlen(cpuarr));

    hipMalloc(&gpuarr, sz);
    hipMemcpy(gpuarr, cpuarr, sz, hipMemcpyHostToDevice);
    
    dkernel<<<1,strlen(cpuarr)>>>(gpuarr, strlen(cpuarr));
    
    hipMemcpy(cpuarr, gpuarr, sz, hipMemcpyDeviceToHost);
    printf(cpuarr);
    
    return 0;
}