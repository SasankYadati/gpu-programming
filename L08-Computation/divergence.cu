
#include <hip/hip_runtime.h>
__global__ void dkernel(unsigned *vector, unsigned vectorsize) {
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    // thread divergence - different warp threads execute different instructions
    if (id % 2) { // odd no. threads
        vector[id] = id;
    }
    else { // even no. threads
        vector[id] = vectorsize * vectorsize;
    }
    vector[id]++;
}