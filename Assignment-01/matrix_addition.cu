#include <iostream>
#include <hip/hip_runtime.h>
#include <kernels.h>
using namespace std;

int main() {
    int m,n;
    cin>>m>>n;
    int *matrixA, *matrixB;
    matrixA = (int*)(malloc(m*n*sizeof(int)));
    matrixB = (int*)(malloc(m*n*sizeof(int)));
    readMatrix(matrixA, m*n);
    readMatrix(matrixB, m*n);
    return 0;
}

void readMatrix(int *matrix, int count) {
    int elem;
    for (int i=0; i<count; i++) {
        cin>>elem;
        matrix[i] = elem;
    }
}