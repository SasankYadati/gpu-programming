#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dkernel() {
    // only one block with ID (0,0,0) and only one thread within a block with ID (0,0,0)
    if (threadIdx.x == 0 && blockIdx.x == 0 &&
        threadIdx.y == 0 && blockIdx.y == 0 &&
        threadIdx.z == 0 && blockIdx.z == 0) {
            printf("%d %d %d %d %d %d \n", gridDim.x, gridDim.y, gridDim.z, blockDim.x, blockDim.y, blockDim.z);
    }
}

int main() {
    dim3 grid(2,3,4);
    dim3 block(5,6,7);
    dkernel<<<grid, block>>>();
    // #threads = 2*3*4*5*6*7
    // #threads in a block = 5*6*7
    // #blocks in the grid for dkernel = 2*3*4
    hipDeviceSynchronize();
    return 0;
}