#include <iostream>
#include <hip/hip_runtime.h>

int main() {
       int deviceCount;
       hipError_t error_id = hipGetDeviceCount(&deviceCount);
       if (error_id != hipSuccess) {
              printf("hipGetDeviceCount returned %d\n-> %s\n",
              static_cast<int>(error_id), hipGetErrorString(error_id));
              printf("Result = FAIL\n");
              exit(EXIT_FAILURE);
       }
       if (deviceCount == 0) {
              printf("There are no available device(s) that support CUDA\n");
       } else {
              printf("Detected %d CUDA Capable device(s)\n", deviceCount);
       }

       for (int dev = 0; dev < deviceCount; ++dev) {
              hipSetDevice(dev);
              hipDeviceProp_t deviceProp;
              hipGetDeviceProperties(&deviceProp, dev);

              printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);
              printf("  Total amount of constant memory:               %zu bytes\n", deviceProp.totalConstMem);
              printf("  GPU Max Clock rate:                            %0.2f GHz\n", deviceProp.clockRate * 1e-6f);
              printf("  Multiprocessors:                               %d Multiprocessors\n", deviceProp.multiProcessorCount);
              printf("  Maximum number of threads per multiprocessor:  %d\n", deviceProp.maxThreadsPerMultiProcessor);
        }
       return 0;
}