#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void init(int *da, int alen) {
    int i = threadIdx.x;
    if (i < alen) {
        da[i] = 0;
    }
}

__global__ void addIndex(int *da, int alen) {
    int i = threadIdx.x;
    if (i < alen) {
        da[i] += i;
    }
}

int main() {
    int *da;
    const int N = 8000;
    hipMalloc(&da, N * sizeof(int));
    
    init<<<1,N>>>(da, N);
    addIndex<<<1,N>>>(da, N);
    hipDeviceSynchronize();

    int a[N];
    hipMemcpy(a, da, N*sizeof(int), hipMemcpyDeviceToHost);

    for (int i=0; i<N; i++) {
        printf("%d ",a[i]);
    }

    return 0;
}