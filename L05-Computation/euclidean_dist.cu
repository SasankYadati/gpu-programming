#include<iostream>
#include<hip/hip_runtime.h>
#include <cmath>
#define BLOCK_SIZE 1024

using namespace std;

__global__ void dkernel(int *vectorX, int *vectorY, float *dist, int N) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int id1, id2;
    id1 = id / N;
    id2 = id % N;
    float dist_;
    if (id<N*N) {
        auto x1 = vectorX[id1], y1 = vectorY[id1], x2 = vectorX[id2], y2 = vectorY[id2];
        dist_ = sqrt((float)(pow(x1 - x2, 2) + pow(y1 - y2, 2)));
        dist[id] = dist_;
    }
}

void readVectors(int *vec1, int *vec2, int count);


int main(int nn, char *str[]) {
    unsigned N = atoi(str[1]);
    unsigned nblocks = ceil((float)(N*N) / BLOCK_SIZE);

    dim3 blockdims(N,1);

    int *vectorX, *vectorY, *hvectorX, *hvectorY;
    
    hipMalloc(&vectorX, N * sizeof(int));
    hipMalloc(&vectorY, N * sizeof(int));
    hvectorX = (int *)(malloc(N*sizeof(int)));
    hvectorY = (int *)(malloc(N*sizeof(int)));
    
    readVectors(hvectorX, hvectorY, N);
    hipMemcpy(vectorX, hvectorX, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(vectorY, hvectorY, N*sizeof(int), hipMemcpyHostToDevice);

    float *dist, *hdist;
    hipMalloc(&dist, N * N * sizeof(float));
    hdist = (float *)(malloc(N * N * sizeof(float)));

    dkernel<<<nblocks,BLOCK_SIZE>>>(vectorX, vectorY, dist, N);
    hipMemcpy(hdist, dist, N * N * sizeof(float), hipMemcpyDeviceToHost);

    for (unsigned ii=0; ii<N; ii+=1) {
        for (unsigned jj=0; jj<N; jj+=1) {
            printf("%6.2f ", hdist[ii * N + jj]);
        }
        printf("\n");
    }

    return 0;
}

void readVectors(int *vec1, int *vec2, int count) {
    int x,y;
    for (int i=0; i<count; i++) {
        cin>>x>>y;;
        vec1[i] = x;
        vec2[i] = y;
    }
}