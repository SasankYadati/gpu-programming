#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "kernels.h"
#define BLOCKSIZE 1024

using namespace std;

void readMatrix(int *matrix, int count);

int main() {
    int m,n;
    cin>>m>>n;
    int *hmatrixA, *hmatrixB, *hmatrixC;
    hmatrixA = (int*)(malloc(m*n*sizeof(int)));
    hmatrixB = (int*)(malloc(m*n*sizeof(int)));
    hmatrixC = (int*)(malloc(m*n*sizeof(int)));
    readMatrix(hmatrixA, m*n);
    readMatrix(hmatrixB, m*n);
    
    int *matrixA, *matrixB, *matrixC;
    hipMalloc(&matrixA, m*n*sizeof(int));
    hipMalloc(&matrixB, m*n*sizeof(int));
    hipMalloc(&matrixC, m*n*sizeof(int));
    
    hipMemcpy(matrixA, hmatrixA, m*n*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(matrixB, hmatrixB, m*n*sizeof(int), hipMemcpyHostToDevice);

    int num_threads_per_block = m < BLOCKSIZE ? m : BLOCKSIZE;
    int num_blocks = ceil((float)m/BLOCKSIZE);

    per_row_kernel<<<num_blocks,num_threads_per_block>>>(m, n, matrixA, matrixB, matrixC);

    hipMemcpy(hmatrixC, matrixC, m*n*sizeof(int), hipMemcpyDeviceToHost);

    for (int i=0; i<m; i++) {
        for (int j=0; j<n; j++) {
            printf("%d ", hmatrixC[i * m + j]);
        }
        printf("\n");
    }

    num_threads_per_block = n < BLOCKSIZE ? n : BLOCKSIZE;
    dim3 blockDims(num_threads_per_block, num_threads_per_block);

    per_column_kernel<<<num_blocks, blockDims>>>(m, n, matrixA, matrixB, matrixC);
    
    return 0;
}

void readMatrix(int *matrix, int count) {
    int elem;
    for (int i=0; i<count; i++) {
        cin>>elem;
        matrix[i] = elem;
    }
}